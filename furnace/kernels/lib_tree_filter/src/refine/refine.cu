#include "hip/hip_runtime.h"
#include <math.h>
#include <thread>
#include <vector>
#include <deque>
#include <iostream>
#include <stdlib.h>
#include <fstream>
#include <hip/hip_runtime.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <THC/THCAtomics.cuh>

#define CUDA_NUM_THREADS         64
#define GET_CUDA_CHANNEL(N)      ceil(512.0f / N)

template <typename scalar_t>
__global__ void root_leaf_prop_kernel(
        scalar_t * in_data, 
        scalar_t * out_data, 
        scalar_t * weight,
        int * sorted_index, 
        int * sorted_parent_index, 
        int batch_size, 
        int channel_size, 
        int vertex_count){

    const int thread_idx    = threadIdx.x;
    const int batch_idx     = blockIdx.x;
    const int channel_idx   = blockIdx.y;
    const int thread_count  = blockDim.x;
    const int channel_step  = gridDim.y;

    in_data             += batch_idx * vertex_count * channel_size;
    out_data            += batch_idx * vertex_count * channel_size;
    weight              += batch_idx * vertex_count;
    sorted_index        += batch_idx * vertex_count;
    sorted_parent_index += batch_idx * vertex_count;

    __shared__ int node_per_thread[CUDA_NUM_THREADS];
    node_per_thread[thread_idx] = -1;
    if (thread_idx == 0){
        weight[0]              = 0;
        sorted_parent_index[0] = 0;
    }
    __syncthreads();

    int i = thread_idx;
    while (i < vertex_count){
        int par = sorted_parent_index[i];
        int par_thread = par % thread_count;
        if ((node_per_thread[par_thread] >= par) || (i == 0)){
            int cur_pos = sorted_index[i];
            int par_pos = sorted_index[par];
            for (int k = channel_idx * vertex_count; k < channel_size * vertex_count;
                       k += channel_step * vertex_count){
                scalar_t edge_weight = weight[i];
                out_data[cur_pos + k] = in_data[i + k] * (1 - edge_weight * edge_weight) +
                                        out_data[par_pos + k] * edge_weight;
                __threadfence_block();
            }
            node_per_thread[thread_idx] = i;
            i += thread_count;
        }
        __syncthreads();
    }
}

template <typename scalar_t>
__global__ void leaf_root_aggr_kernel(
        scalar_t * in_data, 
        scalar_t * out_data, 
        scalar_t * weight,
        int * sorted_index, 
        int * sorted_child_index, 
        int batch_size, 
        int channel_size, 
        int vertex_count,
        int max_adj_per_node){

    const int thread_idx    = threadIdx.x;
    const int batch_idx     = blockIdx.x;
    const int channel_idx   = blockIdx.y;
    const int thread_count  = blockDim.x;
    const int channel_step  = gridDim.y;
    
    if (in_data != NULL){
        in_data    += batch_idx * vertex_count * channel_size;
    }    
    out_data             += batch_idx * vertex_count * channel_size;
    weight               += batch_idx * vertex_count;
    sorted_index         += batch_idx * vertex_count;
    sorted_child_index   += batch_idx * vertex_count * max_adj_per_node;

    __shared__ int node_per_thread[CUDA_NUM_THREADS];
    node_per_thread[thread_idx] = vertex_count;
    __syncthreads();

    int i = vertex_count - thread_idx - 1;
    while (i >= 0){
        int child_len = 0;
        bool valid = true;
        for (int j = 0; j < max_adj_per_node; j++){
            int child        = sorted_child_index[i * max_adj_per_node + j];
            int child_thread = (vertex_count - child - 1) % thread_count;

            if (child <= 0) break;
            if (node_per_thread[child_thread] > child){
                valid = false;
                break;
            }
            child_len++;
        }
        if (valid){
            int cur_pos = sorted_index[i];
            for (int k = channel_idx * vertex_count; k < channel_size * vertex_count; 
                    k += channel_step * vertex_count){
                scalar_t aggr_sum;
                if (in_data != NULL)    
                    aggr_sum = in_data[cur_pos + k];
                else
                    aggr_sum = 1;
                for (int j = 0; j < child_len; j++){
                    int child = sorted_child_index[i * max_adj_per_node + j];
                    aggr_sum += out_data[child + k] * weight[child];
                }
                out_data[i + k] = aggr_sum;
            }
            node_per_thread[thread_idx] = i;
            i -= thread_count;
        }
        __syncthreads();
    }
}

template <typename scalar_t>
__global__ void root_leaf_grad_kernel(
        scalar_t * in_data,
        scalar_t * in_grad,
        scalar_t * out_data,
        scalar_t * out_grad, 
        scalar_t * weight,
        scalar_t * grad,
        int * sorted_index, 
        int * sorted_parent_index, 
        int batch_size, 
        int data_channel_size,
        int grad_channel_size,
        int vertex_count){

    const int thread_idx    = threadIdx.x;
    const int batch_idx     = blockIdx.x;
    const int channel_idx   = blockIdx.y;
    const int thread_count  = blockDim.x;
    const int channel_step  = gridDim.y;
    const int channel_size  = data_channel_size > grad_channel_size ? data_channel_size : grad_channel_size;

    in_data             += batch_idx * vertex_count * data_channel_size;
    in_grad             += batch_idx * vertex_count * grad_channel_size;
    out_data            += batch_idx * vertex_count * data_channel_size;
    out_grad            += batch_idx * vertex_count * grad_channel_size;
    weight              += batch_idx * vertex_count;
    grad                += batch_idx * vertex_count * channel_size;
    sorted_index        += batch_idx * vertex_count;
    sorted_parent_index += batch_idx * vertex_count;

    __shared__ int node_per_thread[CUDA_NUM_THREADS];
    node_per_thread[thread_idx] = -1;

    int i = thread_idx;
    while (i < vertex_count){
        int cur         = i;
        int par         = sorted_parent_index[i];
        int par_pos     = sorted_index[par];
        int par_thread  = par % thread_count;
        if ((cur == 0) || (node_per_thread[par_thread] >= par)){
            for (int k = channel_idx; k < channel_size; k += channel_step){
                scalar_t edge_weight   = weight[i];
                int data_offset     = (k % data_channel_size) * vertex_count;
                int grad_offset     = (k % grad_channel_size) * vertex_count;
                int out_offset      = k * vertex_count;
                
                if (cur > 0){
                    scalar_t left  = in_grad[cur + grad_offset] * (out_data[par_pos + data_offset] - edge_weight * in_data[cur + data_offset]);
                    scalar_t right = in_data[cur + data_offset] * (out_grad[par + grad_offset] - edge_weight * in_grad[cur + grad_offset]);

                    grad[cur + out_offset]      = left + right;
                    out_grad[cur + grad_offset] = in_grad[cur + grad_offset] * (1 - edge_weight * edge_weight) +
                                                  out_grad[par + grad_offset] * edge_weight;
                    __threadfence_block();
                }
                else
                    grad[cur + out_offset] = 0;
            }
            node_per_thread[thread_idx] = i;
            i += thread_count;
        }
        __syncthreads();
    }
}

std::tuple<at::Tensor, at::Tensor, at::Tensor, at::Tensor, at::Tensor>
refine_forward(
        const at::Tensor & feature_in_tensor, 
        const at::Tensor & edge_weight_tensor, 
        const at::Tensor & sorted_index_tensor, 
        const at::Tensor & sorted_parent_tensor, 
        const at::Tensor & sorted_child_tensor 
    ){
    
    const int batch_size        = feature_in_tensor.size(0);
    const int channel_size      = feature_in_tensor.size(1); 
    const int vertex_size       = feature_in_tensor.size(2);
    const int max_adj_per_node  = sorted_child_tensor.size(2);

    auto options                  = feature_in_tensor.options();
    auto feature_aggr_tensor      = at::zeros_like(feature_in_tensor, options);
    auto feature_aggr_up_tensor   = at::zeros_like(feature_in_tensor, options);
    auto weight_sum_tensor        = at::zeros({batch_size, vertex_size}, options);
    auto weight_sum_up_tensor     = at::zeros({batch_size, vertex_size}, options);

    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(feature_in_tensor.scalar_type(), "refine_forward", [&] {
        scalar_t * feature_in          = feature_in_tensor.contiguous().data_ptr<scalar_t>();
        scalar_t * edge_weight         = edge_weight_tensor.contiguous().data_ptr<scalar_t>();
        int * sorted_index             = sorted_index_tensor.contiguous().data_ptr<int>();
        int * sorted_parent_index      = sorted_parent_tensor.contiguous().data_ptr<int>();
        int * sorted_child_index       = sorted_child_tensor.contiguous().data_ptr<int>();
        scalar_t * feature_aggr        = feature_aggr_tensor.contiguous().data_ptr<scalar_t>();
        scalar_t * feature_aggr_sum    = feature_aggr_up_tensor.contiguous().data_ptr<scalar_t>();
        scalar_t * weight_sum          = weight_sum_tensor.contiguous().data_ptr<scalar_t>();
        scalar_t * weight_aggr_sum     = weight_sum_up_tensor.contiguous().data_ptr<scalar_t>();

        dim3 feature_block_dims(CUDA_NUM_THREADS, 1, 1), feature_grid_dims(batch_size, channel_size, 1);
        leaf_root_aggr_kernel <<< feature_grid_dims, feature_block_dims, sizeof(int) * CUDA_NUM_THREADS, stream >>>(
                feature_in, feature_aggr_sum, edge_weight, sorted_index, sorted_child_index, batch_size, channel_size, vertex_size, max_adj_per_node);
        root_leaf_prop_kernel <<< feature_grid_dims, feature_block_dims, sizeof(int) * CUDA_NUM_THREADS, stream >>>(
                feature_aggr_sum, feature_aggr, edge_weight, sorted_index, sorted_parent_index, batch_size, channel_size, vertex_size);

        dim3 weight_block_dims(CUDA_NUM_THREADS, 1, 1), weight_grid_dims(batch_size, 1, 1);
        leaf_root_aggr_kernel <<< weight_grid_dims, weight_block_dims, sizeof(int) * CUDA_NUM_THREADS, stream >>>(
                static_cast<scalar_t *>(NULL), weight_aggr_sum, edge_weight, sorted_index, sorted_child_index, batch_size, 1, vertex_size, max_adj_per_node);
        root_leaf_prop_kernel <<< weight_grid_dims, weight_block_dims, sizeof(int) * CUDA_NUM_THREADS, stream >>>(
                weight_aggr_sum, weight_sum, edge_weight, sorted_index, sorted_parent_index, batch_size, 1, vertex_size);
    });

    auto feature_out_tensor = feature_aggr_tensor / weight_sum_tensor.unsqueeze(1); 
    auto result = std::make_tuple(feature_out_tensor, feature_aggr_tensor, feature_aggr_up_tensor,
            weight_sum_tensor, weight_sum_up_tensor);

    return result;
}

at::Tensor refine_backward_feature(
        const at::Tensor & feature_in_tensor, 
        const at::Tensor & edge_weight_tensor, 
        const at::Tensor & sorted_index_tensor, 
        const at::Tensor & sorted_parent_tensor, 
        const at::Tensor & sorted_child_tensor,
        const at::Tensor & feature_out_tensor,
        const at::Tensor & feature_aggr_tensor,
        const at::Tensor & feature_aggr_up_tensor,
        const at::Tensor & weight_sum_tensor,
        const at::Tensor & weight_sum_up_tensor,
        const at::Tensor & grad_out_tensor
    ){

    auto options                        = feature_in_tensor.options();
    auto grad_feature_tensor            = at::zeros_like(feature_in_tensor, options);
    auto grad_feature_aggr_sum_tensor   = at::zeros_like(feature_in_tensor, options);

    auto grad_out_norm_tensor = grad_out_tensor / weight_sum_tensor.unsqueeze(1);

    const int batch_size        = feature_in_tensor.size(0);
    const int channel_size      = feature_in_tensor.size(1); 
    const int vertex_size       = feature_in_tensor.size(2);
    const int max_adj_per_node  = sorted_child_tensor.size(2);

    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(feature_in_tensor.scalar_type(), "refine_backward_feature", [&] {
        scalar_t * feature_in          = feature_in_tensor.contiguous().data_ptr<scalar_t>();
        scalar_t * edge_weight         = edge_weight_tensor.contiguous().data_ptr<scalar_t>();
        int * sorted_index             = sorted_index_tensor.contiguous().data_ptr<int>();
        int * sorted_parent_index      = sorted_parent_tensor.contiguous().data_ptr<int>();
        int * sorted_child_index       = sorted_child_tensor.contiguous().data_ptr<int>();
        scalar_t * feature_aggr        = feature_aggr_tensor.contiguous().data_ptr<scalar_t>();
        scalar_t * feature_aggr_sum    = feature_aggr_up_tensor.contiguous().data_ptr<scalar_t>();
        scalar_t * weight_sum          = weight_sum_tensor.contiguous().data_ptr<scalar_t>();
        scalar_t * weight_aggr_sum     = weight_sum_up_tensor.contiguous().data_ptr<scalar_t>();
        scalar_t * grad_out            = grad_out_tensor.contiguous().data_ptr<scalar_t>();
        scalar_t * grad_feature        = grad_feature_tensor.contiguous().data_ptr<scalar_t>();

        scalar_t * grad_out_norm           = grad_out_norm_tensor.contiguous().data_ptr<scalar_t>();
        scalar_t * grad_feature_aggr_sum   = grad_feature_aggr_sum_tensor.contiguous().data_ptr<scalar_t>();

        dim3 feature_block_dims(CUDA_NUM_THREADS, 1, 1), feature_grid_dims(batch_size, channel_size, 1);
        leaf_root_aggr_kernel <<< feature_grid_dims, feature_block_dims, sizeof(int) * CUDA_NUM_THREADS, stream >>>(
                grad_out_norm, grad_feature_aggr_sum, edge_weight, sorted_index, sorted_child_index, batch_size, channel_size, vertex_size, max_adj_per_node);
        root_leaf_prop_kernel <<< feature_grid_dims, feature_block_dims, sizeof(int) * CUDA_NUM_THREADS, stream >>>(
                grad_feature_aggr_sum, grad_feature, edge_weight, sorted_index, sorted_parent_index, batch_size, channel_size, vertex_size);
    });

    return grad_feature_tensor;
}

at::Tensor refine_backward_weight(
        const at::Tensor & feature_in_tensor, 
        const at::Tensor & edge_weight_tensor, 
        const at::Tensor & sorted_index_tensor, 
        const at::Tensor & sorted_parent_tensor, 
        const at::Tensor & sorted_child_tensor,
        const at::Tensor & feature_out_tensor,
        const at::Tensor & feature_aggr_tensor,
        const at::Tensor & feature_aggr_up_tensor,
        const at::Tensor & weight_sum_tensor,
        const at::Tensor & weight_sum_up_tensor,
        const at::Tensor & grad_out_tensor
    ){

    auto options            = feature_in_tensor.options();
    auto grad_weight_tensor = at::zeros_like(edge_weight_tensor, options);

    const int batch_size        = feature_in_tensor.size(0);
    const int channel_size      = feature_in_tensor.size(1); 
    const int vertex_size       = feature_in_tensor.size(2);
    const int max_adj_per_node  = sorted_child_tensor.size(2);
        
    auto grad_all_channel_tensor        = at::zeros_like(feature_in_tensor, options);
    auto grad_norm_all_channel_tensor   = at::zeros_like(feature_in_tensor, options);
    auto grad_out_norm_aggr_sum_tensor  = at::zeros_like(feature_in_tensor, options);
    auto feature_grad_aggr_sum_tensor   = at::zeros_like(feature_in_tensor, options);

    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(feature_in_tensor.scalar_type(), "refine_backward_weight", [&] {
        scalar_t * feature_in          = feature_in_tensor.contiguous().data_ptr<scalar_t>();
        scalar_t * edge_weight         = edge_weight_tensor.contiguous().data_ptr<scalar_t>();
        int * sorted_index             = sorted_index_tensor.contiguous().data_ptr<int>();
        int * sorted_parent_index      = sorted_parent_tensor.contiguous().data_ptr<int>();
        int * sorted_child_index       = sorted_child_tensor.contiguous().data_ptr<int>();
        scalar_t * feature_out         = feature_out_tensor.contiguous().data_ptr<scalar_t>();
        scalar_t * feature_aggr        = feature_aggr_tensor.contiguous().data_ptr<scalar_t>();
        scalar_t * feature_aggr_sum    = feature_aggr_up_tensor.contiguous().data_ptr<scalar_t>();
        scalar_t * weight_sum          = weight_sum_tensor.contiguous().data_ptr<scalar_t>();
        scalar_t * weight_aggr_sum     = weight_sum_up_tensor.contiguous().data_ptr<scalar_t>();
        scalar_t * grad_out            = grad_out_tensor.contiguous().data_ptr<scalar_t>();
        scalar_t * grad_weight         = grad_weight_tensor.contiguous().data_ptr<scalar_t>();
        
        scalar_t * grad_all_channel            = grad_all_channel_tensor.contiguous().data_ptr<scalar_t>();
        scalar_t * grad_norm_all_channel       = grad_norm_all_channel_tensor.contiguous().data_ptr<scalar_t>();
        scalar_t * grad_out_norm_aggr_sum      = grad_out_norm_aggr_sum_tensor.contiguous().data_ptr<scalar_t>();
        scalar_t * feature_grad_aggr_sum       = feature_grad_aggr_sum_tensor.contiguous().data_ptr<scalar_t>();

        auto grad_out_norm_tensor = grad_out_tensor / weight_sum_tensor.unsqueeze(1);
        auto feature_grad_tensor  = grad_out_norm_tensor * feature_out_tensor; 
        scalar_t * grad_out_norm     = grad_out_norm_tensor.contiguous().data_ptr<scalar_t>();
        scalar_t * feature_grad      = feature_grad_tensor.contiguous().data_ptr<scalar_t>();

        dim3 feature_block_dims(CUDA_NUM_THREADS, 1, 1), feature_grid_dims(batch_size, channel_size, 1);
        leaf_root_aggr_kernel <<< feature_grid_dims, feature_block_dims, sizeof(int) * CUDA_NUM_THREADS, stream >>>(
                grad_out_norm, grad_out_norm_aggr_sum, edge_weight, sorted_index, sorted_child_index, batch_size, channel_size, vertex_size, max_adj_per_node);
        leaf_root_aggr_kernel <<< feature_grid_dims, feature_block_dims, sizeof(int) * CUDA_NUM_THREADS, stream >>>(
                feature_grad, feature_grad_aggr_sum, edge_weight, sorted_index, sorted_child_index, batch_size, channel_size, vertex_size, max_adj_per_node);

        root_leaf_grad_kernel <<< feature_grid_dims, feature_block_dims, sizeof(int) * CUDA_NUM_THREADS, stream >>>(
                feature_aggr_sum, grad_out_norm_aggr_sum, feature_aggr, grad_out_norm_aggr_sum, edge_weight, grad_all_channel, 
                sorted_index, sorted_parent_index, batch_size, channel_size, channel_size, vertex_size);
        root_leaf_grad_kernel <<< feature_grid_dims, feature_block_dims, sizeof(int) * CUDA_NUM_THREADS, stream >>>(
                weight_aggr_sum, feature_grad_aggr_sum, weight_sum, feature_grad_aggr_sum, edge_weight, grad_norm_all_channel, 
                sorted_index, sorted_parent_index, batch_size, 1, channel_size, vertex_size);

    });

    grad_weight_tensor = (grad_all_channel_tensor - grad_norm_all_channel_tensor).sum(1);

    return grad_weight_tensor;
}
